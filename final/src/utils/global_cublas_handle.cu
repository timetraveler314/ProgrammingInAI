//
// Created by timetraveler314 on 1/19/25.
//

#include "global_cublas_handle.cuh"

#include <hipblas.h>

hipblasHandle_t &global_cublas_handle::get_instance() {
    static hipblasHandle_t instance;
    static bool initialized = false;

    if (!initialized) {
        hipblasCreate(&instance);
        initialized = true;
    }

    return instance;
}
