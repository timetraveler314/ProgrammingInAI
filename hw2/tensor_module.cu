#include <pybind11/pybind11.h>
#include <pybind11/stl.h>
#include <pybind11/numpy.h>
#include "tensor.h"
#include "tensornn.cuh"

namespace py = pybind11;

py::array_t<float> numpy(const Tensor &t) {
    py::array_t<float> np(t.getShape());
    float *np_data = np.mutable_data();
    hipMemcpy(np_data, t.getRawData(), t.size() * sizeof(float),
               t.getDevice() == TensorDevice::CPU ? hipMemcpyHostToHost
                                                : hipMemcpyDeviceToHost);
    return np;
}

Tensor from_numpy(py::array_t<float, py::array::c_style | py::array::forcecast> np) {
    std::vector<int> shape(np.shape(), np.shape() + np.ndim());
    // By default, we put the tensor on the CPU
    Tensor t(shape, TensorDevice::GPU);
    hipMemcpy(t.getRawData(), np.data(), t.size() * sizeof(float),
               t.getDevice() == TensorDevice::CPU ? hipMemcpyHostToHost
                                               : hipMemcpyHostToDevice);
    return t;
}

PYBIND11_MODULE(Genshin, m) {
    py::enum_<TensorDevice>(m, "TensorDevice")
        .value("CPU", TensorDevice::CPU)
        .value("GPU", TensorDevice::GPU)
        .export_values();

    py::class_<Tensor>(m, "Tensor", py::buffer_protocol())
        .def("numpy", &numpy)
        .def_static("from_numpy", &from_numpy)
        .def(py::init<const std::vector<int>&, TensorDevice>(),
             py::arg("shape"), py::arg("device"))
        .def_static("iota", &Tensor::iota)
        .def("shape", &Tensor::getShape)
        .def("__repr__", [](const Tensor &t) {
            std::string device = t.getDevice() == TensorDevice::CPU ? "CPU" : "GPU";
            std::string shape = "[";
            for (int i = 0; i < t.getShape().size(); i++) {
                shape += std::to_string(t.getShape()[i]);
                if (i != t.getShape().size() - 1) {
                    shape += ", ";
                }
            }
            shape += "]";
            return "<Tensor shape=" + shape + " on " + device + ">";
        })
        .def("__str__", &Tensor::toString);
        // .def("__add__", &operator+)
        // .def("__sub__", &operator-)
        // .def("__mul__", &operator*)
        // .def("__truediv__", &operator/);

    // TensorNN namespace
    py::module nn = m.def_submodule("nn");
    nn.def("forward_fc", &TensorNN::forward_fc);
    nn.def("backward_fc", &TensorNN::backward_fc);
    nn.def("forward_softmax", &TensorNN::forward_softmax);
}