#include <iostream>

#include "tensor.h"
#include "tensor_kernel.h"
#include <thrust/functional.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>

#include "global_curand_generator.cuh"
#include "tensornn.cuh"

Tensor get_test_neg1_1_tensor();

int main() {
    auto x = Tensor({2, 3}, TensorDevice::CPU);
    auto w = Tensor({4, 3}, TensorDevice::CPU);
    auto b = Tensor({4}, TensorDevice::CPU);

    auto rand_tensor1 = Tensor::uniform({2,3,4}, TensorDevice::CPU);

    std::cout << rand_tensor1 << std::endl;

    for (int i = 0; i < x.size(); i++) {
        x.data->space[i] = i;
    }
    for (int i = 0; i < w.size(); i++) {
        w.data->space[i] = i;
    }
    for (int i = 0; i < b.size(); i++) {
       b.data->space[i] = i;
    }

    auto xg = x.gpu();
    auto wg = w.gpu();
    auto bg = b.gpu();

    std::cout << "X: " << x << std::endl;
    std::cout << "W: " << w << std::endl;
    std::cout << "B: " << b << std::endl;

    auto result = TensorNN::forward_fc(xg, wg, bg);

    std::cout << "Result: " << result << std::endl;

    auto [dx, dw, db] = TensorNN::backward_fc(get_test_neg1_1_tensor(), xg, wg);

    std::cout << "dx: " << dx << std::endl;
    std::cout << "dw: " << dw << std::endl;
    std::cout << "db: " << db << std::endl;

    auto rand_tensor = Tensor::uniform({2,3,4}, TensorDevice::GPU);

    std::cout << rand_tensor << std::endl;

    return 0;
}

Tensor random_gpu_tensor(const std::vector<int>& shape) {
    Tensor t(shape, TensorDevice::CPU);


    return t.gpu();
}

Tensor get_test_neg1_1_tensor() {
    Tensor t1({2,4}, TensorDevice::CPU);

    for (int i = 0; i < t1.size(); i++) {
        t1.data->space[i] = i % 2 == 0 ? -1 : 1;
    }

    return t1.gpu();
}
