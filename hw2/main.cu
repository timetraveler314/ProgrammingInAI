#include <iostream>

#include "tensor.h"
#include "tensor_kernel.h"
#include <thrust/functional.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>

#include "global_curand_generator.cuh"
#include "tensornn.cuh"

Tensor get_test_neg1_1_tensor();

int main() {
    auto x = Tensor::iota({2, 3}, TensorDevice::GPU);
    auto w = Tensor::iota({4, 3}, TensorDevice::GPU);
    auto b = Tensor::iota({4}, TensorDevice::GPU);

    std::cout << "X: " << x << std::endl;
    std::cout << "W: " << w << std::endl;
    std::cout << "B: " << b << std::endl;

    auto result = TensorNN::forward_fc(x, w, b);

    std::cout << "Result: " << result << std::endl;

    auto [dx, dw, db] = TensorNN::backward_fc(get_test_neg1_1_tensor(), x, w);

    std::cout << "dx: " << dx << std::endl;
    std::cout << "dw: " << dw << std::endl;
    std::cout << "db: " << db << std::endl;

    auto rand_tensor = Tensor::uniform({2,3,4}, TensorDevice::GPU);

    std::cout << rand_tensor << std::endl;

    return 0;
}

Tensor random_gpu_tensor(const std::vector<int>& shape) {
    Tensor t(shape, TensorDevice::CPU);


    return t.gpu();
}

Tensor get_test_neg1_1_tensor() {
    Tensor t1({2,4}, TensorDevice::CPU);

    for (int i = 0; i < t1.size(); i++) {
        t1.getRawData()[i] = i % 2 == 0 ? -1 : 1;
    }

    return t1.gpu();
}
