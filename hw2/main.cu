#include <iostream>

#include "tensor.h"
#include "tensor_kernel.h"
#include <thrust/functional.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>

#include "tensornn.cuh"

Tensor random_gpu_tensor(const std::vector<int>& shape);
Tensor get_test_neg1_1_tensor();

int main() {
    srand(42); // seed random number generator to allow for reproducing the results.

    auto x = Tensor({2, 3}, TensorDevice::CPU);
    auto w = Tensor({4, 3}, TensorDevice::CPU);
    auto b = Tensor({4}, TensorDevice::CPU);

    for (int i = 0; i < x.size(); i++) {
        x.data->space[i] = i;
    }
    for (int i = 0; i < w.size(); i++) {
        w.data->space[i] = i;
    }
    for (int i = 0; i < b.size(); i++) {
       b.data->space[i] = i;
    }

    auto xg = x.gpu();
    auto wg = w.gpu();
    auto bg = b.gpu();

    std::cout << "X: " << x << std::endl;
    std::cout << "W: " << w << std::endl;
    std::cout << "B: " << b << std::endl;

    auto result = TensorNN::forward_fc(xg, wg, bg);

    std::cout << "Result: " << result << std::endl;

    auto [dx, dw, db] = TensorNN::backward_fc(get_test_neg1_1_tensor(), xg, wg);

    std::cout << "dx: " << dx << std::endl;
    std::cout << "dw: " << dw << std::endl;
    std::cout << "db: " << db << std::endl;

    return 0;
}

Tensor random_gpu_tensor(const std::vector<int>& shape) {
    Tensor t(shape, TensorDevice::CPU);


    return t.gpu();
}

Tensor get_test_neg1_1_tensor() {
    Tensor t1({2,4}, TensorDevice::CPU);

    for (int i = 0; i < t1.size(); i++) {
        t1.data->space[i] = i % 2 == 0 ? -1 : 1;
    }

    return t1.gpu();
}
